#include "hip/hip_runtime.h"
/*
Keep P2 state permanently

This utility helps overclocked systems not to enter P0 state and crash.

usage: cudaidle [device=N]

Contact petri33 @ setiathome

nvcc -O3 -I ./cuda-samples/Common cudaidle.cu -o cudaidle
*/

#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// a variable in GPU memory
__device__ int i;

__global__ void kernel(int val)
{
	i = 0;
}

int main(int argc, char **argv)
{
	int devID;
	hipDeviceProp_t props;

	// This will pick selected or the best possible CUDA capable device
	devID = findCudaDevice(argc, (const char **)argv);

	//Get GPU information
	checkCudaErrors(hipGetDevice(&devID));
	checkCudaErrors(hipGetDeviceProperties(&props, devID));
	printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
	printf("Keep in P2 state enabled.\nCreated 2018 by petri33 @ setiathome\n\n");

	//minimal Kernel configuration
	dim3 dimGrid(1);
	dim3 dimBlock(1);

	unsigned int microseconds = 100000; // 0.1 seconds

	for(;;)
	{
		// run 10 times a second, negligible performance hit
		kernel<<<dimGrid, dimBlock>>>(0);
		usleep(microseconds);
	}

	return EXIT_SUCCESS;
}
